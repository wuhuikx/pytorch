#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void bitwise_xor_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    // Boolean type does not work with ^ (bitwise XOR) in C++. bitwise_xor wraps this operation for both Boolean and
    // integral types.
    gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(bool a, bool b) {
            return a != b;
          });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_xor_cuda", [&]() {
      gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
            return a ^ b;
      });
    });
  }
}

void logical_xor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.common_dtype(), "logical_xor_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return static_cast<scalar_t>(bool(a) != bool(b));
    });
  });
}

void smooth_l1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto z = fabs(a - b);
      return z < scalar_t(1.) ? scalar_t(0.5) * z * z : z - scalar_t(0.5);
    });
  });
}

void sigmoid_backward_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "sigmoid_backward_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a * (scalar_t(1.) - b) * b;
    });
  });
}

void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(bitwise_xor_stub, &bitwise_xor_kernel_cuda);
REGISTER_DISPATCH(logical_xor_stub, &logical_xor_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(sigmoid_backward_stub, &sigmoid_backward_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

}} // namespace at::native
