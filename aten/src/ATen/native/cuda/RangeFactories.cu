#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/HIPContext.h>
#include <cmath>
#include <limits>

#include <ATen/native/cuda/Loops.cuh>

namespace at {
namespace native {

Tensor& linspace_cuda_out(Tensor& result, Scalar start, Scalar end, int64_t steps) {
  TORCH_CHECK(steps >= 0, "number of steps must be non-negative");

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  // Using TensorIter, output no longer need to be contiguous
  // We still need to check if there is internal overlap
  // YES: error out, TOO_HARD: fallback to copy behavior, NO: use result directly
  auto overlap = has_internal_overlap(result);
  TORCH_CHECK(overlap != MemOverlap::YES,
              "unsupported operation: more than one element of the written-to tensor "
              "refers to a single memory location. Please clone() the tensor before "
              "performing the operation.");
  Tensor r = (overlap == MemOverlap::TOO_HARD) ?  at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(start);
  } else if (isIntegralType(r.scalar_type(), 0)) {
    AT_DISPATCH_INTEGRAL_TYPES(r.scalar_type(), "linspace_cuda", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      float step = static_cast<float>(scalar_end - scalar_start) / (steps - 1);

      auto iter = TensorIterator::nullary_op(r);
      gpu_kernel_with_index(iter, [scalar_start, step]GPU_LAMBDA(int ind) -> scalar_t {
        scalar_t val = scalar_start + step * ind;
        return val;
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(r.scalar_type(), "linspace_cuda", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);

      auto iter = TensorIterator::nullary_op(r);
      gpu_kernel_with_index(iter, [scalar_start, step]GPU_LAMBDA(int ind) -> scalar_t {
        scalar_t val = scalar_start + step * ind;
        return val;
      });
    });
  }

  if(overlap == MemOverlap::TOO_HARD) {
    result.copy_(r);
  }

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& logspace_cuda_out(Tensor& result, Scalar start, Scalar end, int64_t steps, double base) {
  TORCH_CHECK(steps >= 0, "number of steps must be non-negative");

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  // Using TensorIter, output no longer need to be contiguous
  // We still need to check if there is internal overlap
  // YES: error out, TOO_HARD: fallback to copy behavior, NO: use result directly
  auto overlap = has_internal_overlap(result);
  TORCH_CHECK(overlap != MemOverlap::YES,
              "unsupported operation: more than one element of the written-to tensor "
              "refers to a single memory location. Please clone() the tensor before "
              "performing the operation.");
  Tensor r = (overlap == MemOverlap::TOO_HARD) ?  at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(std::pow(base, start.to<double>()));
  } else if (isIntegralType(r.scalar_type(), 0)) {
    AT_DISPATCH_INTEGRAL_TYPES(r.scalar_type(), "logspace_cuda", [&]() {
      float scalar_base = static_cast<float>(base); // Use float to avoid promotion to double
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      float step = static_cast<float>(scalar_end - scalar_start) / (steps - 1);

      auto iter = TensorIterator::nullary_op(r);
      gpu_kernel_with_index(iter, [scalar_start, step, scalar_base]GPU_LAMBDA(int ind) -> scalar_t {
        scalar_t val = std::pow(scalar_base, scalar_start + step * ind);
        return val;
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(r.scalar_type(), "logspace_cuda", [&]() {
      scalar_t scalar_base = static_cast<scalar_t>(base);
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);

      auto iter = TensorIterator::nullary_op(r);
      gpu_kernel_with_index(iter, [scalar_start, step, scalar_base]GPU_LAMBDA(int ind) -> scalar_t {
          scalar_t val = std::pow(scalar_base, scalar_start + step * ind);
          return val;
        });
    });
  }

  if(overlap == MemOverlap::TOO_HARD) {
    result.copy_(r);
  }

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& range_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, result.scalar_type(), "range_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    TORCH_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    TORCH_CHECK(std::isfinite(static_cast<double>(xstart)) &&
             std::isfinite(static_cast<double>(xend)),
             "unsupported range: ", xstart, " -> ", xend);
    TORCH_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
             "upper bound and larger bound inconsistent with step sign");
    int64_t size = static_cast<int64_t>(((xend - xstart) / xstep) + 1);

    if (result.numel() != size) {
      result.resize_({size});
    }
    // Using TensorIter, output no longer need to be contiguous
    // We still need to check if there is internal overlap
    // YES: error out, TOO_HARD: fallback to copy behavior, NO: use result directly
    auto overlap = has_internal_overlap(result);
    TORCH_CHECK(overlap != MemOverlap::YES,
                "unsupported operation: more than one element of the written-to tensor "
                "refers to a single memory location. Please clone() the tensor before "
                "performing the operation.");
    Tensor r = (overlap == MemOverlap::TOO_HARD) ?  at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

    auto iter = TensorIterator::nullary_op(r);
    gpu_kernel_with_index(iter, [xstart, xstep]GPU_LAMBDA(int ind) -> scalar_t {
        accscalar_t inc = xstep * static_cast<accscalar_t>(ind);
        accscalar_t val = xstart + inc;
        return static_cast<scalar_t>(val);
    });

    if(overlap == MemOverlap::TOO_HARD) {
      result.copy_(r);
    }

  });

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& arange_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, result.scalar_type(), "arange_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    // we use double precision for (start - end) / step
    // to compute size_d for consistency across devices.
    // The problem with using accscalar_t is that accscalar_t might be float32 on gpu for a float32 scalar_t,
    // but double on cpu for the same,
    // and the effective output size starts differing on CPU vs GPU because of precision issues, which
    // we dont want.
    // the corner-case we do want to take into account is int64_t, which has higher precision than double
    double size_d;
    if (std::is_same<scalar_t, int64_t>::value) {
      size_d = std::ceil(static_cast<double>(end.to<accscalar_t>() - start.to<accscalar_t>())
                         / step.to<accscalar_t>());
    } else {
      size_d = std::ceil(static_cast<double>(end.to<double>() - start.to<double>())
                         / step.to<double>());
    }

    TORCH_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    TORCH_CHECK(std::isfinite(static_cast<double>(xstart)) &&
             std::isfinite(static_cast<double>(xend)),
             "unsupported range: ", xstart, " -> ", xend);
    TORCH_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
             "upper bound and larger bound inconsistent with step sign");

    TORCH_CHECK(size_d >= 0 && size_d <= static_cast<double>(std::numeric_limits<int64_t>::max()),
             "invalid size, possible overflow?");
    int64_t size = static_cast<int64_t>(size_d);
    int64_t numel = result.numel();

    if (numel != size) {
      if(numel > 0){
        TORCH_WARN("The number of elements in the out tensor of shape ", result.sizes(),
                    " is ", numel, " which does not match the computed number of elements ", size,
                    ". Note that this may occur as a result of rounding error. "
                    "The out tensor will be resized to a tensor of shape (", size, ",).");
      }
      result.resize_({size});
    }
    // Using TensorIter, output no longer need to be contiguous
    // We still need to check if there is internal overlap
    // YES: error out, TOO_HARD: fallback to copy behavior, NO: use result directly
    auto overlap = has_internal_overlap(result);
    TORCH_CHECK(overlap != MemOverlap::YES,
                "unsupported operation: more than one element of the written-to tensor "
                "refers to a single memory location. Please clone() the tensor before "
                "performing the operation.");
    Tensor r = (overlap == MemOverlap::TOO_HARD) ?  at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

    auto iter = TensorIterator::nullary_op(r);
    gpu_kernel_with_index(iter, [xstart, xstep]GPU_LAMBDA(int ind) -> scalar_t {
        accscalar_t inc = xstep * static_cast<accscalar_t>(ind);
        accscalar_t val = xstart + inc;
        return static_cast<scalar_t>(val);
    });

    if(overlap == MemOverlap::TOO_HARD) {
      result.copy_(r);
    }

  });

  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

}} // namespace at::native
