#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

constexpr float EPSILON = 1e-12;

namespace {

using namespace at;

template<typename scalar_t>
void kl_div_backward_kernel(const Tensor& grad_input, const Tensor& target, const Tensor& grad) {
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
      grad_input,
      target,
      grad,
      [] __device__(
          scalar_t& grad_input_val, const scalar_t& target_val, const scalar_t& grad_val) {
        if (target_val > 0) {
          grad_input_val = -target_val * grad_val;
        }
      });
}

template<typename scalar_t>
void binary_cross_entropy_out_kernel(Tensor& loss, const Tensor& input, const Tensor& target) {
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
    loss,
    input,
    target,
    [] __device__(
      scalar_t& loss_val,
      const scalar_t& input_val,
      const scalar_t& target_val
    ) {
      const scalar_t zero = 0;
      const scalar_t one = 1;
      const scalar_t neg_100 = -100;

      CUDA_KERNEL_ASSERT(input_val >= zero && input_val <= one);

      scalar_t log_input_val = log(input_val);
      scalar_t log_1_minus_input_val = log(one - input_val);

      log_input_val = max(log_input_val, neg_100);
      log_1_minus_input_val = max(log_1_minus_input_val, neg_100);

      loss_val = ((target_val - one) * log_1_minus_input_val) - (target_val * log_input_val);
    }
  );
}

template<typename scalar_t>
void binary_cross_entropy_backward_out_kernel(Tensor& grad_input, const Tensor& grad, const Tensor& input, const Tensor& target) {
  at::cuda::CUDA_tensor_apply4<scalar_t, scalar_t, scalar_t, scalar_t>(
    grad_input,
    grad,
    input,
    target,
    [] __device__(
      scalar_t& grad_input_val,
      const scalar_t& grad_val,
      const scalar_t& input_val,
      const scalar_t& target_val
    ) {
      const scalar_t one = 1;
      const scalar_t epsilon = EPSILON;

      scalar_t grad_input_denominator = max(
        (one - input_val) * input_val,
        epsilon
      );

      grad_input_val = grad_val * (input_val - target_val) / grad_input_denominator;
    }
  );
}

} // namespace

namespace at { namespace native {

Tensor kl_div_backward_cuda(const Tensor& grad, const Tensor& input, const Tensor& target, int64_t reduction) {
  auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor grad_expand = grad.expand_as(input);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "kl_div_backward_cuda", [&]() {
    kl_div_backward_kernel<scalar_t>(grad_input, target, grad_expand);
  });
  if (reduction == at::Reduction::Mean) {
    return grad_input / input.numel();
  }
  return grad_input;
}

Tensor binary_cross_entropy_cuda(const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
    Tensor loss = at::empty_like(input);
    return at::native::binary_cross_entropy_out_cuda(loss, input, target, weight, reduction);
}

Tensor& binary_cross_entropy_out_cuda(Tensor& loss, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "binary_cross_entropy_out_cuda", [&]() {
    binary_cross_entropy_out_kernel<scalar_t>(loss, input, target);
  });
  if (weight.defined()) {
    loss.mul_(weight);
  }

  if (reduction != at::Reduction::None) {
    Tensor loss_reduced;
    if (reduction == at::Reduction::Mean) {
      loss_reduced = loss.mean();
    } else if (reduction == at::Reduction::Sum) {
      loss_reduced = loss.sum();
    }
    loss.resize_as_(loss_reduced).copy_(loss_reduced);
  }

  return loss;
}

Tensor binary_cross_entropy_backward_cuda(const Tensor& grad, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  Tensor grad_input = at::empty_like(input);
  return at::native::binary_cross_entropy_backward_out_cuda(grad_input, grad, input, target, weight, reduction);
}

Tensor& binary_cross_entropy_backward_out_cuda(Tensor& grad_input, const Tensor& grad, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  Tensor grad_expand = grad.expand_as(input);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "binary_cross_entropy_backward_out_cuda", [&]() {
    binary_cross_entropy_backward_out_kernel<scalar_t>(grad_input, grad_expand, input, target);
  });

  if (weight.defined()) {
    grad_input.mul_(weight);
  }
  if (reduction == at::Reduction::Mean) {
    grad_input.div_(input.numel());
  }
  return grad_input;
}

}}  // namespace at::native
